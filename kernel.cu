#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <thrust\device_vector.h>
#include <thrust\reduce.h>
#include <stdio.h>
#include <ctime>
#include <string>
#include <fstream>
#include <map>

// Parses a string and stores data into a vector of vector of strings
void parse(std::string& someString, std::vector<std::vector<std::string>>& attributeTable)
{
    int attributeCount = 0;
    std::vector<std::string> vectorOfStrings;
    while (someString.length() != 0 && someString.find(',') != std::string::npos)
    {
        size_t pos;
        std::string singleAttribute;
        pos = someString.find_first_of(',');
        singleAttribute = someString.substr(0, pos);
        vectorOfStrings.push_back(singleAttribute);
        someString.erase(0, pos + 1);
    }
    vectorOfStrings.push_back(someString);
    attributeTable.push_back(vectorOfStrings);
    vectorOfStrings.clear();
}

// class node to store a binary tree node
class Node
{
public:
    int attr_index;
    float attr_value;
    bool isleaf;
    int l;
public:
    __host__ __device__
    Node()
    {
        attr_index = 0;
        attr_value = 0;
        isleaf = false;
        l = 0;
    }
    __host__ __device__
    Node(int index, float value, bool leaf, int lef)
    {
        attr_index = index;
        attr_value = value;
        isleaf = leaf;
        l = lef;
    }
    __host__ __device__
    bool is_leaf()
    {
        return isleaf;
    }
    __host__ __device__
    bool def_left()
    {
        return true;
    }
    __host__ __device__
    int fid()
    {
        return attr_index;
    }
    __host__ __device__
    int left(int val)
    {
        return l;
    }
    __host__ __device__
    float thresh()
    {
        return attr_value;
    }
};

// class tree to store a array of binary tree nodes. Has the rensponsibility to transfer the nodes to gpu
class Tree
{
public:
    Node* host_node_array;
    Node* dev_node_array;
    int size;
    __host__ __device__
    Tree(Node* node_array, int s)
    {
        host_node_array = node_array;
        dev_node_array = 0;
        size = s;
    }

    __device__
    Node& operator[](int index)
    {
        assert(index < size);
        return dev_node_array[index];
    }

    __host__
    void AssignNodesToGpu()
    {
        hipError_t cudaStatus = hipMalloc((void**)&dev_node_array, size * sizeof(Node));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
        }
        cudaStatus = hipMemcpy(dev_node_array, host_node_array, size * sizeof(Node), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
        }

    }

};

// device code to infer one tree
__device__ float InferOneTree(Tree tree, const float* input)
{
    int curr = 0;
    int count = 0;
 
    for (;;) {
        Node n = tree[curr];
        if (n.is_leaf()) break;
        float val = input[n.fid()];
        bool cond = isnan(val) ? !n.def_left() : val >= n.thresh();
        curr = n.left(curr) + cond;
        count++;
    }

    float out = tree[curr].thresh();

    return out;
}

// kernel to infer one tree
__global__ void MySingleTreeKernel(Tree* tree, float* input, int columns)
{
    int no_of_columns = columns;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // slice the input
    float* input_slice = input + i * no_of_columns;
    float output = InferOneTree(*tree, input_slice);
    //printf("output %f", output);
}



int main()
{
    /*const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };*/
    using namespace std;
    clock_t begin = clock();

    const int treeSize = 31;

    Node treenodes[treeSize];

    // create a binary decision tree.
    treenodes[0] = Node(7,0.052,false,1);

    treenodes[1] = Node (20,16.54,false,3);
    treenodes[2] = Node (26,0.225,false,5);

    treenodes[3] = Node(13,37.61,false,7);
    treenodes[4] = Node(21,20.22,false,9);

    treenodes[5] = Node(-1,2.0,true,-1);
    treenodes[6] = Node(23,710.2,false,11);

    treenodes[7] = Node(21,33.27,false,13);
    treenodes[8] = Node(4,0.091,false,15);

    treenodes[9] = Node(-1,2.0,true,-1);
    treenodes[10] = Node(17,0.011,false,17);

    treenodes[11] = Node(21,25.95,false,19);
    treenodes[12] = Node(1,14.12,false,21);

    treenodes[13] = Node(-1,2.0,true,-1);
    treenodes[14] = Node(21,34.14,false,23);

    treenodes[15] = Node(-1,2.0,true,-1);
    treenodes[16] = Node(17,0.012,false,25);

    treenodes[17] = Node(-1,1.0,true,-1);
    treenodes[18] = Node(-1,2.0,true,-1);

    treenodes[19] = Node(-1,2.0,true,-1);
    treenodes[20] = Node(9,0.065,false,27);

    treenodes[21] = Node(25,0.361,false,29);
    treenodes[22] = Node(-1,1.0,true,-1);

    treenodes[23] = Node(-1,1.0,true,-1);
    treenodes[24] = Node(-1,2.0,true,-1);

    treenodes[25] = Node(-1,2.0,true,-1);
    treenodes[26] = Node(-1,1.0,true,-1);

    treenodes[27] = Node(-1,2.0,true,-1);
    treenodes[28] = Node(-1,1.0,true,-1);

    treenodes[29] = Node(-1,1.0,true,-1);
    treenodes[30] = Node(-1,2.0,true,-1);



    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    Tree my_tree = Tree(treenodes, treeSize);

    my_tree.AssignNodesToGpu();

    Tree* dev_my_tree;

    // Assign memory to device tree
    cudaStatus = hipMalloc((void**)&dev_my_tree, sizeof(Tree));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    // Copy tree from host memory to GPU.
    cudaStatus = hipMemcpy(dev_my_tree, &my_tree,  sizeof(Tree), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    //// Create input dataset
    //const int no_of_input = 1048576;
    //float* host_input = new float[2*no_of_input];
    //float* dev_input = 0;
    //for (int i = 0; i < 2 * no_of_input; i++)
    //{
    //    host_input[i] = i % 50;
    //    i++;
    //    host_input[i] = i % 20;

    //}

    // read input file
    map<string, float> mresult;
    mresult["B"] = 1.0;
    mresult["M"] = 2.0;

    ifstream inputFile;// Input file stream
    string singleInstance;// Single line read from the input file 
    vector<vector<string>> dataTable;// Input data in the form of a vector of vector of strings
    vector<vector<float>> dataTableDouble;// Input data in the form of a vector of vector of floats
    inputFile.clear();
    inputFile.open("data.csv"); // Open test file
    if (!inputFile) // Exit if test file is not found
    {
        cerr << "Error: Testing data file not found!" << endl;
        exit(-1);
    }
    while (getline(inputFile, singleInstance)) // Store test data in a table
    {
        parse(singleInstance, dataTable);
    }

    int row = dataTable.size() - 1;
    int column = dataTable[0].size() - 1;

    const int no_of_input = 569;
    const int no_of_columns = 32;
   
    const int simulate_blocks = 1;

    float* dataArrayFloat = new float[simulate_blocks * no_of_input * no_of_columns];

    

    // Stores the predicted class labels for each row in Int
    vector<float> predictedClassLabelsfloat;
    // Stores the given class labels in the test data in Int
    vector<float> givenClassLabelsfloat;

    // Store given class labels in vector of strings named givenClassLabelsDouble
    // Transfer input data from string to Int using map
    for (int i = 1; i < dataTable.size(); i++)
    {
        string data = dataTable[i][1];
        float dataFloat = mresult[data];
        givenClassLabelsfloat.push_back(dataFloat);
        for (int j = 2; j < dataTable[0].size() - 1; j++) {
            dataArrayFloat[(i - 1)*32 + j-1] = std::stof(dataTable[i][j]);
        }
    }

    float* dev_input = 0;

    for (int i = 0; i < simulate_blocks; i++)
    {
        for (int j = 0; j < 569 * 32; j++)
        {
            dataArrayFloat[i*569*32 + j] = dataArrayFloat[j];
        }
    }

    cudaStatus = hipMalloc((void**)&dev_input, simulate_blocks * no_of_columns * no_of_input * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_input, dataArrayFloat, simulate_blocks * no_of_columns * no_of_input * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventRecord(start, 0);

    MySingleTreeKernel << <simulate_blocks, 569 >> >(dev_my_tree, dev_input, no_of_columns);

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed kernel time : %f ms\n", elapsedTime);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
       
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
    }

    clock_t end = clock();
    double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    printf("Elapsed CPU time : %f ms\n", elapsed_secs*1000);

    
    

    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);

    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    ////cudaStatus = InferWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //printf("%d %d %d %d %d",c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// 
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
